#include "hip/hip_runtime.h"
#include "../interface/PulseChiSqSNNLS.h"
#include <math.h>
#include <iostream>
#include <thrust/swap.h>


// __global__ void GpuDoFit(PulseChiSqSNNLS *pulse, DoFitArgs *parameters, bool *status){
  //   int i = blockIdx.x*blockDim.x + threadIdx.x;
  //   auto args = parameters[i];
  //   status[i] = pulse[i].DoFit(args.samples, args.samplecor, args.pederr, args.bxs, args.fullpulse, args.fullpulsecov);
  // }


__host__ __device__ bool PulseChiSqSNNLS::DoFit(const SampleVector &samples, const SampleMatrix &samplecor, 
                                       double pederr, const BXVector &bxs, const FullSampleVector &fullpulse,
                                       const FullSampleMatrix &fullpulsecov) {
  
  const unsigned int nsample = SampleVector::RowsAtCompileTime;
  const unsigned int npulse = bxs.rows();
  
  _sampvec = samples;
  _bxs = bxs;
  
  _pulsemat = SamplePulseMatrix::Zero(nsample,npulse);
  _ampvec = PulseVector::Zero(npulse);
  _errvec = PulseVector::Zero(npulse);  
  _nP = 0;
  _chisq = 0.;
  
  //initialize pulse template matrix
  for (unsigned int ipulse=0; ipulse<npulse; ++ipulse) {
    int bx = _bxs.coeff(ipulse);
    int firstsamplet = std::max(0,bx + 3);
    int offset = 7-3-bx;
    
    const unsigned int nsamplepulse = nsample-firstsamplet;
    _pulsemat.col(ipulse).segment(firstsamplet,nsamplepulse) = fullpulse.segment(firstsamplet+offset,nsamplepulse);
  }
  
  //do the actual fit
  bool status = Minimize(samplecor,pederr,fullpulsecov);
  _ampvecmin = _ampvec;
  
//   std::cout << " _sampvec = " << _sampvec << std::endl;
//   std::cout << " bxs = " << bxs << std::endl;
//   std::cout << " fullpulse = " << fullpulse << std::endl;
//   std::cout << " _ampvecmin = " << _ampvecmin << std::endl;
  
  _bxsmin = _bxs;
  
  if (!status) return status;
  
//   std::cout << " _computeErrors = " << _computeErrors << std::endl;
  
  if(!_computeErrors) return status;
  
  //compute MINOS-like uncertainties for in-time amplitude
  bool foundintime = false;
  unsigned int ipulseintime = 0;
//   std::cout << " npulse = " << npulse << std::endl;
  for (unsigned int ipulse=0; ipulse<npulse; ++ipulse) {
//     std::cout << " _bxs.coeff( " << ipulse << "::" << npulse << " ) = " << _bxs.coeff(ipulse) << std::endl;
    if (_bxs.coeff(ipulse)==0) {
      ipulseintime = ipulse;
      foundintime = true;
      break;
    }
  }
//   std::cout << " foundintime = " << foundintime << std::endl;
  if (!foundintime) return status;
  
  
  
  const unsigned int ipulseintimemin = ipulseintime;
  
  double approxerr = ComputeApproxUncertainty(ipulseintime);
  double chisq0 = _chisq;
  double x0 = _ampvecmin[ipulseintime];  
  
  //move in time pulse first to active set if necessary
  if (ipulseintime<_nP) {
    _pulsemat.col(_nP-1).swap(_pulsemat.col(ipulseintime));
    thrust::swap(_ampvec.coeffRef(_nP-1),_ampvec.coeffRef(ipulseintime));
    thrust::swap(_bxs.coeffRef(_nP-1),_bxs.coeffRef(ipulseintime));
    ipulseintime = _nP - 1;
    --_nP;    
  }
  
  
  
  SampleVector pulseintime = _pulsemat.col(ipulseintime);
  _pulsemat.col(ipulseintime).setZero();
  
  //two point interpolation for upper uncertainty when amplitude is away from boundary
  double xplus100 = x0 + approxerr;
  _ampvec.coeffRef(ipulseintime) = xplus100;
  _sampvec = samples - _ampvec.coeff(ipulseintime)*pulseintime;  

//   std::cout << " here 1 " << std::endl;
  status &= Minimize(samplecor,pederr,fullpulsecov);
  if (!status) return status;
  double chisqplus100 = ComputeChiSq();
  
//   std::cout << " here 2 " << std::endl;
  
  
  double sigmaplus = std::abs(xplus100-x0)/sqrt(chisqplus100-chisq0);
  
  //if amplitude is sufficiently far from the boundary, compute also the lower uncertainty and average them
  if ( (x0/sigmaplus) > 0.5 ) {
    for (unsigned int ipulse=0; ipulse<npulse; ++ipulse) {
      if (_bxs.coeff(ipulse)==0) {
        ipulseintime = ipulse;
        break;
      }
    }    
    double xminus100 = std::max(0.,x0-approxerr);
    _ampvec.coeffRef(ipulseintime) = xminus100;
    _sampvec = samples - _ampvec.coeff(ipulseintime)*pulseintime;
    status &= Minimize(samplecor,pederr,fullpulsecov);
    if (!status) return status;
    double chisqminus100 = ComputeChiSq();
    
    double sigmaminus = std::abs(xminus100-x0)/sqrt(chisqminus100-chisq0);
    _errvec[ipulseintimemin] = 0.5*(sigmaplus + sigmaminus);
    
  }
  else {
    _errvec[ipulseintimemin] = sigmaplus;
  }
  
  _chisq = chisq0;  
  
  return status;
  
}

__host__ __device__ bool PulseChiSqSNNLS::Minimize(const SampleMatrix &samplecor, double pederr, 
                                          const FullSampleMatrix &fullpulsecov) {
  
  
  const int maxiter = 50;
  for (int iter=0; iter<maxiter; ++iter){
    if(!(updateCov(samplecor,pederr,fullpulsecov) &&  NNLS()))
      return false;    
    double chisqnow = ComputeChiSq();
    double deltachisq = chisqnow-_chisq; 
    _chisq = chisqnow;
    if (std::abs(deltachisq)<1e-3)
      break;
  }    
  return true;  
}

__host__ __device__ bool PulseChiSqSNNLS::updateCov(const SampleMatrix &samplecor, double pederr,
                                           const FullSampleMatrix &fullpulsecov) {
  const unsigned int nsample = SampleVector::RowsAtCompileTime;
  const unsigned int npulse = _bxs.rows();
  
  _invcov.triangularView<Eigen::Lower>() = (pederr*pederr)*samplecor;
  
  for (unsigned int ipulse=0; ipulse<npulse; ++ipulse) {
    if (_ampvec.coeff(ipulse)==0.) continue;
    int bx = _bxs.coeff(ipulse);
    int firstsamplet = std::max(0,bx + 3);
    int offset = 7-3-bx;
    
    double ampsq = _ampvec.coeff(ipulse)*_ampvec.coeff(ipulse);
    
    const unsigned int nsamplepulse = nsample-firstsamplet;    
    _invcov.block(firstsamplet,firstsamplet,nsamplepulse,nsamplepulse).triangularView<Eigen::Lower>() +=
      ampsq*fullpulsecov.block(firstsamplet+offset,firstsamplet+offset,nsamplepulse,nsamplepulse);    
  }
  
  _covdecomp.compute(_invcov);
    
  return true;  
}

__host__ __device__ double PulseChiSqSNNLS::ComputeChiSq() {
  
  //   SampleVector resvec = _pulsemat*_ampvec - _sampvec;
  //   return resvec.transpose()*_covdecomp.solve(resvec);
  
  // TODO: port Eigen::LLT solve to gpu
  return _covdecomp.matrixL().solve(_pulsemat*_ampvec - _sampvec).squaredNorm();
  // return 1.0;
}

__host__ __device__ double PulseChiSqSNNLS::ComputeApproxUncertainty(unsigned int ipulse) {
  //compute approximate uncertainties
  //(using 1/second derivative since full Hessian is not meaningful in
  //presence of positive amplitude boundaries.)
   

  // TODO: port Eigen::LLT solve to gpu
  return 1./_covdecomp.matrixL().solve(_pulsemat.col(ipulse)).norm();
  // return 1.;
  
}

__host__ __device__ bool PulseChiSqSNNLS::NNLS() {
  
  //Fast NNLS (fnnls) algorithm as per http://citeseerx.ist.psu.edu/viewdoc/download?doi=10.1.1.157.9203&rep=rep1&type=pdf
  
  const unsigned int npulse = _bxs.rows();
  // TODO: Port EigenLLT to gpu
  SamplePulseMatrix invcovp = _covdecomp.matrixL().solve(_pulsemat);
  // SamplePulseMatrix invcovp;

  PulseMatrix aTamat(npulse,npulse);
  aTamat.triangularView<Eigen::Lower>() = invcovp.transpose()*invcovp;
  aTamat = aTamat.selfadjointView<Eigen::Lower>();

  // TODO: Port EigenLLT to gpu
  PulseVector aTbvec = invcovp.transpose()*_covdecomp.matrixL().solve(_sampvec);  
  // PulseVector aTbvec;
  
  PulseVector wvec(npulse);
  
  
  for (int iter=0; iter<1000; iter++) {    
    //can only perform this step if solution is guaranteed viable
    if (iter>0 || _nP==0) {
      if ( _nP==npulse ) break;                  
      
      const unsigned int nActive = npulse - _nP;
      
      wvec.tail(nActive) = aTbvec.tail(nActive) - (aTamat.selfadjointView<Eigen::Lower>()*_ampvec).tail(nActive);       
      
      Index idxwmax;
      double wmax = wvec.tail(nActive).maxCoeff(&idxwmax);
      
      //convergence
      if (wmax<1e-11) break;
      
      //unconstrain parameter
      Index idxp = _nP + idxwmax;
      //printf("adding index %i, orig index %i\n",int(idxp),int(_bxs.coeff(idxp)));
      aTamat.col(_nP).swap(aTamat.col(idxp));
      aTamat.row(_nP).swap(aTamat.row(idxp));
      _pulsemat.col(_nP).swap(_pulsemat.col(idxp));
      thrust::swap(aTbvec.coeffRef(_nP),aTbvec.coeffRef(idxp));
      thrust::swap(_ampvec.coeffRef(_nP),_ampvec.coeffRef(idxp));
      thrust::swap(_bxs.coeffRef(_nP),_bxs.coeffRef(idxp));
      ++_nP;
    }
    
    
    while (_nP > 0) {
      //printf("iter in, idxsP = %i\n",int(_idxsP.size()));
      
//       std::cout << " >>  iter = " << iter << std::endl;
      
      // TODO: port EigenLDLT solve to gpu
      PulseVector ampvecpermtest = _ampvec;
      
      //solve for unconstrained parameters      
      
      // TODO: port Eigen::LDLT solve to gpu
      ampvecpermtest.head(_nP) = aTamat.topLeftCorner(_nP,_nP).ldlt().solve(aTbvec.head(_nP));     
      // ampvecpermtest.head(_nP) = aTamat.topLeftCorner(_nP,_nP);     
     
      //check solution
      if (ampvecpermtest.head(_nP).minCoeff()>0.) {
        _ampvec.head(_nP) = ampvecpermtest.head(_nP);
        break;
      }      
      
      //update parameter vector
      Index minratioidx=0;
      
      double minratio = std::numeric_limits<double>::max();
      for (unsigned int ipulse=0; ipulse<_nP; ++ipulse) {
        if (ampvecpermtest.coeff(ipulse)<=0.) {
          double ratio = _ampvec.coeff(ipulse)/(_ampvec.coeff(ipulse)-ampvecpermtest.coeff(ipulse));
          if (ratio<minratio) {
            minratio = ratio;
            minratioidx = ipulse;
          }
        }
      }
      
      _ampvec.head(_nP) += minratio*(ampvecpermtest.head(_nP) - _ampvec.head(_nP));
      
      //avoid numerical problems with later ==0. check
      _ampvec.coeffRef(minratioidx) = 0.;
      
      //printf("removing index %i, orig idx %i\n",int(minratioidx),int(_bxs.coeff(minratioidx)));
      aTamat.col(_nP-1).swap(aTamat.col(minratioidx));
      aTamat.row(_nP-1).swap(aTamat.row(minratioidx));
      _pulsemat.col(_nP-1).swap(_pulsemat.col(minratioidx));
      thrust::swap(aTbvec.coeffRef(_nP-1),aTbvec.coeffRef(minratioidx));
      thrust::swap(_ampvec.coeffRef(_nP-1),_ampvec.coeffRef(minratioidx));
      thrust::swap(_bxs.coeffRef(_nP-1),_bxs.coeffRef(minratioidx));
      --_nP;
      
    }
  }
  
  return true;
  
  
}

__host__ __device__ PulseChiSqSNNLS::PulseChiSqSNNLS() : _chisq(0.), _computeErrors(true) {}

__global__ void kernel_multifit(DoFitArgs *vargs, Output *vresults, unsigned int n) {
    // thread idx
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=n) return;

    PulseChiSqSNNLS pulse;
    pulse.disableErrorCalculation();
    auto args = vargs[i];

    // perform the regression
    auto status = pulse.DoFit(args.samples, args.samplecor, args.pederr, args.bxs, args.fullpulse, args.fullpulsecov);

    unsigned int ip_in_time = 0;
    for (unsigned int ip=0; ip<pulse.BXs().rows(); ++ip) {
        if (ip < pulse.BXs().coeff(ip) == 0) {
            ip_in_time = ip;
            break;
        }
    }

    // assing the result
    BXVector BXs_results = pulse.BXs();
    PulseVector X_results = pulse.X();
    
    vresults[i] = Output{pulse.ChiSq(), status ? pulse.X()[ip_in_time] : 0.0, status, BXs_results, X_results};

    // assing the result
    //vresults[i] = DoFitResults{pulse.ChiSq(), pulse.BXs(), pulse.X(), (bool) status}; 
}
