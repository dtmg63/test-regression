#include "../interface/PulseChiSqSNNLS.h"
#include "../interface/PulseChiSqSNNLSWrapper.h"

#include <iostream>
#include <string>
#include <vector>

std::vector<Output> doFitWrapper(std::vector<DoFitArgs> const& vargs) {
  // input parameters to the multifit on gpu
  DoFitArgs* d_args;
  Output* d_results;
  std::vector<Output> results;
  std::cout << "vargs.size() = " << vargs.size() << std::endl;
  results.resize(vargs.size());
  std::cout << "size = " << results.size() << std::endl;
  std::cout << "capacity = " << results.capacity() << std::endl;

  // allocate on the device
  std::cout << "allocate on the device" << std::endl;
  hipMalloc((void**)&d_args, sizeof(DoFitArgs) * vargs.size());
  hipMalloc((void**)&d_results, sizeof(Output) * vargs.size());

  // transfer to the device
  std::cout << "copy to the device " << std::endl;
  hipMemcpy(d_args, vargs.data(), sizeof(DoFitArgs) * vargs.size(),
             hipMemcpyHostToDevice);

  // kernel invoacation
  std::cout << "launch the kenrel" << std::endl;
  int nthreadsPerBlock = 256;
  int nblocks = (vargs.size() + nthreadsPerBlock - 1) / nthreadsPerBlock;
  kernel_multifit<<<nblocks, nthreadsPerBlock>>>(d_args, d_results,
                                                 vargs.size());
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  std::string name = "multifit_gpu";
  if (err != hipSuccess) {
    std::cout << "cuda error!" << std::endl
              << hipGetErrorString(err) << std::endl;
    std::cout << "test " << name << " failed" << std::endl;
  }

  // copy results back
  std::cout << "copy back to the host" << std::endl;
  hipMemcpy(&(results[0]), d_results, sizeof(Output) * results.size(),
             hipMemcpyDeviceToHost);
  std::cout << "vresults.size() = " << results.size() << std::endl;

  // free resources
  std::cout << "free the device memory" << std::endl;
  hipFree(d_args);
  hipFree(d_results);

  std::cout << "vresults.size() [still inside] = " << results.size() << std::endl;
  
  return results;
}
