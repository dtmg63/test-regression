#include "hip/hip_runtime.h"
#include <math.h>
#include <thrust/swap.h>
#include <iostream>
#include "../interface/PulseChiSqSNNLS.h"
#include "nnls/interface/data_types.h"
#include "nnls/interface/inplace_fnnls.h"

__host__ __device__ bool PulseChiSqSNNLS::DoFit(
    const SampleVector& samples,
    const SampleMatrix& samplecor,
    double pederr,
    const BXVector& bxs,
    const FullSampleVector& fullpulse,
    const FullSampleMatrix& fullpulsecov) {
  const unsigned int nsample = SampleVector::RowsAtCompileTime;
  const unsigned int npulse = bxs.rows();

  _sampvec = samples;
  _bxs = bxs;

  _pulsemat = SamplePulseMatrix::Zero(nsample, npulse);
  _ampvec = PulseVector::Zero(npulse);
  _errvec = PulseVector::Zero(npulse);
  _nP = 0;
  _chisq = 0.;

  // initialize pulse template matrix
  for (unsigned int ipulse = 0; ipulse < npulse; ++ipulse) {
    int bx = _bxs.coeff(ipulse);
    int firstsamplet = std::max(0, bx + 3);
    int offset = 7 - 3 - bx;

    const unsigned int nsamplepulse = nsample - firstsamplet;
    _pulsemat.col(ipulse).segment(firstsamplet, nsamplepulse) =
        fullpulse.segment(firstsamplet + offset, nsamplepulse);
  }

  // do the actual fit
  bool status = Minimize(samplecor, pederr, fullpulsecov);
  _ampvecmin = _ampvec;

  //   std::cout << " _sampvec = " << _sampvec << std::endl;
  //   std::cout << " bxs = " << bxs << std::endl;
  //   std::cout << " fullpulse = " << fullpulse << std::endl;
  //   std::cout << " _ampvecmin = " << _ampvecmin << std::endl;

  _bxsmin = _bxs;

  if (!status)
    return status;

  //   std::cout << " _computeErrors = " << _computeErrors << std::endl;

  if (!_computeErrors)
    return status;

  // compute MINOS-like uncertainties for in-time amplitude
  bool foundintime = false;
  unsigned int ipulseintime = 0;
  //   std::cout << " npulse = " << npulse << std::endl;
  for (unsigned int ipulse = 0; ipulse < npulse; ++ipulse) {
    //     std::cout << " _bxs.coeff( " << ipulse << "::" << npulse << " ) = "
    //     << _bxs.coeff(ipulse) << std::endl;
    if (_bxs.coeff(ipulse) == 0) {
      ipulseintime = ipulse;
      foundintime = true;
      break;
    }
  }
  //   std::cout << " foundintime = " << foundintime << std::endl;
  if (!foundintime)
    return status;

  const unsigned int ipulseintimemin = ipulseintime;

  double approxerr = ComputeApproxUncertainty(ipulseintime);
  double chisq0 = _chisq;
  double x0 = _ampvecmin[ipulseintime];

  // move in time pulse first to active set if necessary
  if (ipulseintime < _nP) {
    _pulsemat.col(_nP - 1).swap(_pulsemat.col(ipulseintime));
    thrust::swap(_ampvec.coeffRef(_nP - 1), _ampvec.coeffRef(ipulseintime));
    thrust::swap(_bxs.coeffRef(_nP - 1), _bxs.coeffRef(ipulseintime));
    ipulseintime = _nP - 1;
    --_nP;
  }

  SampleVector pulseintime = _pulsemat.col(ipulseintime);
  _pulsemat.col(ipulseintime).setZero();

  // two point interpolation for upper uncertainty when amplitude is away from
  // boundary
  double xplus100 = x0 + approxerr;
  _ampvec.coeffRef(ipulseintime) = xplus100;
  _sampvec = samples - _ampvec.coeff(ipulseintime) * pulseintime;

  //   std::cout << " here 1 " << std::endl;
  status &= Minimize(samplecor, pederr, fullpulsecov);
  if (!status)
    return status;
  double chisqplus100 = ComputeChiSq();

  //   std::cout << " here 2 " << std::endl;

  double sigmaplus = std::abs(xplus100 - x0) / sqrt(chisqplus100 - chisq0);

  // if amplitude is sufficiently far from the boundary, compute also the lower
  // uncertainty and average them
  if ((x0 / sigmaplus) > 0.5) {
    for (unsigned int ipulse = 0; ipulse < npulse; ++ipulse) {
      if (_bxs.coeff(ipulse) == 0) {
        ipulseintime = ipulse;
        break;
      }
    }
    double xminus100 = std::max(0., x0 - approxerr);
    _ampvec.coeffRef(ipulseintime) = xminus100;
    _sampvec = samples - _ampvec.coeff(ipulseintime) * pulseintime;
    status &= Minimize(samplecor, pederr, fullpulsecov);
    if (!status)
      return status;
    double chisqminus100 = ComputeChiSq();

    double sigmaminus = std::abs(xminus100 - x0) / sqrt(chisqminus100 - chisq0);
    _errvec[ipulseintimemin] = 0.5 * (sigmaplus + sigmaminus);

  } else {
    _errvec[ipulseintimemin] = sigmaplus;
  }

  _chisq = chisq0;

  return status;
}

__host__ __device__ bool PulseChiSqSNNLS::Minimize(
    const SampleMatrix& samplecor,
    double pederr,
    const FullSampleMatrix& fullpulsecov) {
  const int maxiter = 50;
  for (int iter = 0; iter < maxiter; ++iter) {
    if (!(updateCov(samplecor, pederr, fullpulsecov) && NNLS()))
      return false;
    double chisqnow = ComputeChiSq();
    double deltachisq = chisqnow - _chisq;
    _chisq = chisqnow;
    if (std::abs(deltachisq) < 1e-3)
      break;
  }
  return true;
}

__host__ __device__ bool PulseChiSqSNNLS::updateCov(
    const SampleMatrix& samplecor,
    double pederr,
    const FullSampleMatrix& fullpulsecov) {
  const unsigned int nsample = SampleVector::RowsAtCompileTime;
  const unsigned int npulse = _bxs.rows();

  _invcov.triangularView<Eigen::Lower>() = (pederr * pederr) * samplecor;

  for (unsigned int ipulse = 0; ipulse < npulse; ++ipulse) {
    if (_ampvec.coeff(ipulse) == 0.)
      continue;
    int bx = _bxs.coeff(ipulse);
    int firstsamplet = std::max(0, bx + 3);
    int offset = 7 - 3 - bx;

    double ampsq = _ampvec.coeff(ipulse) * _ampvec.coeff(ipulse);

    const unsigned int nsamplepulse = nsample - firstsamplet;
    _invcov.block(firstsamplet, firstsamplet, nsamplepulse, nsamplepulse)
        .triangularView<Eigen::Lower>() +=
        ampsq * fullpulsecov.block(firstsamplet + offset, firstsamplet + offset,
                                   nsamplepulse, nsamplepulse);
  }

  _covdecomp.compute(_invcov);

  return true;
}

__host__ __device__ double PulseChiSqSNNLS::ComputeChiSq() {
  //   SampleVector resvec = _pulsemat*_ampvec - _sampvec;
  //   return resvec.transpose()*_covdecomp.solve(resvec);

  // TODO: port Eigen::LLT solve to gpu
  return _covdecomp.matrixL()
      .solve(_pulsemat * _ampvec - _sampvec)
      .squaredNorm();
  // return 1.0;
}

__host__ __device__ double PulseChiSqSNNLS::ComputeApproxUncertainty(
    unsigned int ipulse) {
  // compute approximate uncertainties
  //(using 1/second derivative since full Hessian is not meaningful in
  // presence of positive amplitude boundaries.)

  // TODO: port Eigen::LLT solve to gpu
  return 1. / _covdecomp.matrixL().solve(_pulsemat.col(ipulse)).norm();
  // return 1.;
}

__host__ __device__ bool PulseChiSqSNNLS::NNLS() {
  FixedMatrix A = _covdecomp.matrixL().solve(_pulsemat);
  FixedVector b = _covdecomp.matrixL().solve(_sampvec);

  // std::cout << A << std::endl;
  // std::cout << b << std::endl;

  // TODO: this should be a parameter not a magic number
  auto const epsilon = 1e-11;
  auto const max_iter = 1000;
  FixedVector x = FixedVector(_ampvec);
  inplace_fnnls(A, b, x, epsilon, max_iter);

  _ampvec = x;

  return true;
}

__host__ __device__ PulseChiSqSNNLS::PulseChiSqSNNLS()
    : _chisq(0.), _computeErrors(true) {}

__global__ void kernel_multifit(DoFitArgs* vargs,
                                Output* vresults,
                                unsigned int n) {
  // thread idx
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;

  PulseChiSqSNNLS pulse;
  pulse.disableErrorCalculation();
  auto args = vargs[i];

  // perform the regression
  auto status = pulse.DoFit(args.samples, args.samplecor, args.pederr, args.bxs,
                            args.fullpulse, args.fullpulsecov);

  unsigned int ip_in_time = 0;
  for (unsigned int ip = 0; ip < pulse.BXs().rows(); ++ip) {
    if (ip < pulse.BXs().coeff(ip) == 0) {
      ip_in_time = ip;
      break;
    }
  }

  
  //---- save all reconstructed amplitudes
//   std::vector<double> v_ampl;
//   for (unsigned int ip=0; ip<pulse.BXs().rows(); ++ip) {
//     v_ampl.push_back(0.);
//   }
//   
//   for (unsigned int ip=0; ip<pulse.BXs().rows(); ++ip) {
//     v_ampl[ (int(pulse.BXs().coeff(ip))) + 5] = (pulse.X())[ ip ];
//   }
  
  // assing the result
//   vresults[i] = Output{pulse.ChiSq(), status ? pulse.X()[ip_in_time] : 0.0, status, v_ampl};
  vresults[i] = Output{pulse.ChiSq(), status ? pulse.X()[ip_in_time] : 0.0, status, pulse.BXs(), pulse.X()};
  
  // assing the result
  // vresults[i] = DoFitResults{pulse.ChiSq(), pulse.BXs(), pulse.X(), (bool)
  // status};
}
